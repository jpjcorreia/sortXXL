#include "hip/hip_runtime.h"
/**
 * @file module_generate_dataset.cu
 * @brief source code file for generating a data set of random numbers
 * @date 2012-12-25 File creation
 * @author Cláudio Esperança <2120917@my.ipleiria.pt>, Diogo Serra <2120915@my.ipleiria.pt>, João Correia <2111415@my.ipleiria.pt>
 */
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hiprand.h>

#include "../3rd/HandleError.h"
#include "../3rd/sortXXL_cmd.h"

#include "constants.h"
#include "module_generate_dataset.h"

/**
 * @brief provides an error management facilitator for CURAND calls
 */
#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)

int generate_dataset_from_arguments(struct gengetopt_args_info args_info, int *data){
	int minimum = 0, maximum = INT_MAX;

	if (args_info.min_given == 1){
		minimum = args_info.min_arg;
	}

	if (args_info.max_given == 1){
		maximum = args_info.max_arg;
	}

	if(args_info.random_given == 1 && args_info.random_arg>0){
		return generate_dataset_in_interval(data, args_info.random_arg, minimum, maximum);
	}
	return EXIT_FAILURE;
}

int generate_dataset(int *data, int n){
	return generate_dataset_in_interval(data, n, 0, INT_MAX);
}

// based on http://docs.nvidia.com/cuda/hiprand/index.html#topic_1_3_1 and http://aresio.blogspot.pt/2011/05/cuda-random-numbers-inside-kernels.html
int generate_dataset_in_interval(int *data, int n, int minimum, int maximum){
	int i;
	hiprandGenerator_t gen;
	float *devData, *hostData;

	/* Allocate n floats on host */
	hostData = (float *)calloc(n, sizeof(float));

	/* Allocate n floats on device */
	HANDLE_ERROR(hipMalloc((void **)&devData, n*sizeof(float)));

	/* Create pseudo-random number generator */
	CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));

	/* Set seed */// added the time as seed
	CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL)));

	/* Generate n floats on device */
	CURAND_CALL(hiprandGenerateUniform(gen, devData, n));

	/* Copy device memory to host */
	HANDLE_ERROR(hipMemcpy(hostData, devData, n * sizeof(float), hipMemcpyDeviceToHost));

	/* Copy the result */// enforced the limits
	for(i = 0; i < n; i++) {
		data[i] = (int) round((minimum + hostData[i] * (maximum-minimum)));
	}

	/* Cleanup */
	CURAND_CALL(hiprandDestroyGenerator(gen));
	HANDLE_ERROR(hipFree(devData));
	free(hostData);
	return EXIT_SUCCESS;
}
